#include "hip/hip_runtime.h"
/* -*-C-*-
 *
 * Copyright Paul Kurucz 2007 - All Rights Reserved
 * For use in distributed.net projects only.
 * Any other distribution or use of this source violates copyright.
 *
 * With modifications by Greg Childers
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ccoreio.h"

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

/* Uncomment the define below to display the    */
/* processing timestamps.  (Linux Only)         */
//#define DISPLAY_TIMESTAMPS

#ifdef DISPLAY_TIMESTAMPS
#include <sys/time.h>
#endif

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

#define P 0xB7E15163
#define Q 0x9E3779B9

#ifdef __cplusplus
extern "C" s32 CDECL rc5_72_unit_func_cuda_2( RC5_72UnitWork *, u32 *, void * );
#endif

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------              Local Variables               --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */


/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------     Local Helper Function Prototypes       --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

/* Type decaration for the L0 field of the      */
/* RC5_72UnitWork structure.                    */
typedef struct {
        u32 hi;
        u32 mid;
        u32 lo;
} L0_t;

static __host__ __device__ u32 swap_u32(u32 num);
static __host__ __device__ u8 add_u32(u32 num1, u32 num2, u32 * result);
static __host__ __device__ void increment_L0(u32 * hi, u32 * mid, u32 * lo, u32 amount);

static __global__ void cuda_2pipe(const u32 plain_hi, const u32 plain_lo,
                                 const u32 cypher_hi, const u32 cypher_lo,
                                 const u32 L0_hi, const u32 L0_mid, const u32 L0_lo,
                                 const u32 process_amount, u8 * results, u8 * match_found);

static s32 CDECL rc5_72_run_cuda_2(RC5_72UnitWork *rc5_72unitwork, u32 *iterations, int device, u32 num_threads, int /*waitmode*/);

#ifdef DISPLAY_TIMESTAMPS
static __inline int64_t linux_read_counter(void);
#endif

#define SHL(x, s) ((u32) ((x) << ((s) & 31)))
#define SHR(x, s) ((u32) ((x) >> (32 - ((s) & 31)))) 

#define ROTL(x, s) ((u32) (SHL((x), (s)) | SHR((x), (s))))
#define ROTL3(x) ROTL(x, 3)

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------           Local Helper Functions           --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

/* u32 byte swap */
static __host__ __device__ u32 swap_u32(u32 num)
{
	u32 retval = (num & 0xFF000000) >> 24;
	retval |= (num & 0x00FF0000) >> 8;
	retval |= (num & 0x0000FF00) << 8;
	retval |= (num & 0x000000FF) << 24;

	return retval;
}

/* Adds two u32s, returning the carry out bit.  */
static __host__ __device__ u8 add_u32(u32 num1, u32 num2, u32 * result)
{
	u8 carry = 0;
	u32 temp = num1;

	temp += num2;

	/* Check for an overflow */
	if(temp < num1) {
		carry = 1;
	}

	/* Pass back the result */
	*result = temp;

	return carry;
}

/* Increments the hi, mid and lo parts of the   */
/* L0 by the specified amount.                  */
static __host__ __device__ void increment_L0(u32 * hi, u32 * mid, u32 * lo, u32 amount)
{
	u32 temp;
	u32 result;
	u8 carry;

	/* Low uint32 */
	temp = *hi & 0xFF;
	temp |= swap_u32(*mid) << 8;
	carry = add_u32(temp, amount, &result);
	*hi = result & 0xFF;
	*mid &= 0x000000FF;
	*mid |= swap_u32(result >> 8);

	/* Mid uint32 */
	if(carry) {
		temp = *mid & 0xFF;
		temp |= swap_u32(*lo) << 8;
		carry = add_u32(temp, 1, &result);
		*mid &= 0xFFFFFF00;
		*mid |= result & 0xFF;
		*lo &= 0x000000FF;
		*lo |= swap_u32(result >> 8);
	}

	if(carry) {
		temp = *lo & 0xFF;
		carry = add_u32(temp, 1, &result);
		*lo &= 0xFFFFFF00;
		*lo |= result & 0xFF;
	}
}




/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------              Core Entry Point              --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

s32 CDECL rc5_72_unit_func_cuda_2(RC5_72UnitWork *rc5_72unitwork, u32 *iterations, void * /*memblk*/)
{
	/* The number of GPU threads per thread block   */
	/* to execute.  The default value of 64 makes   */
	/* optimum usage of __shared__ multiprocessor   */
	/* memory.  The maximum value is 512.           */
	const u32 num_threads = 64;

	return rc5_72_run_cuda_2(rc5_72unitwork, iterations, rc5_72unitwork->threadnum, num_threads, -1);
}

static s32 CDECL rc5_72_run_cuda_2(RC5_72UnitWork *rc5_72unitwork, u32 *iterations, int device, u32 num_threads, int /*waitmode*/)
{
	int currentdevice;
	u32 i;
	u32 grid_dim;
	s32 retval = RESULT_NOTHING;

	/* Local and GPU variable pairs */
	u8 match_found;
	u8 * cuda_match_found = NULL;
	u8 * results = NULL;
	u8 * cuda_results = NULL;

#ifdef DISPLAY_TIMESTAMPS
	int64_t current_ts;
	int64_t prev_ts;
#endif

//	fprintf(stderr, "\r\nRC5 cuda: iterations=%i\r\n", *iterations);
        
	if( hipGetDevice(&currentdevice) != (hipError_t) hipSuccess ) {
		retval = -1;
		fprintf(stderr, "RC5 cuda: ERROR: hipGetDevice\r\n");
		goto error_exit;
	}

	if (currentdevice != device) {
		if( hipSetDevice(device) != (hipError_t) hipSuccess ) {
			retval = -1;
			fprintf(stderr, "RC5 cuda: ERROR: hipSetDevice\r\n");
			goto error_exit;
		}
	}

	/* Determine the grid dimensionality based on the */
	/* number of iterations.                          */
	grid_dim = (*iterations/2 + num_threads - 1) / num_threads;
	if(grid_dim > 65535) {
		grid_dim = 65535;
	}

	/* --------------------------------------------- */

	/* Allocate the cuda_match_found variable */
	if( hipMalloc((void **)&cuda_match_found, sizeof(u8)) != (hipError_t) hipSuccess ) {
		retval = -1;
		fprintf(stderr, "RC5 cuda: ERROR: hipMalloc: cuda_match_found\r\n");
		goto error_exit;
	}

	/* Allocate the results arrays */
	results = (u8 *)malloc(grid_dim * num_threads * 2 * sizeof(u8));
	if( results == NULL ) {
		retval = -1;
		fprintf(stderr, "RC5 cuda: ERROR: malloc\r\n");
		goto error_exit;
	}

	if( hipMalloc((void **)&cuda_results, grid_dim * num_threads * 2 * sizeof(u8)) != (hipError_t) hipSuccess ) {
		retval = -1;
		fprintf(stderr, "RC5 cuda: ERROR: hipMalloc: cuda_results\r\n");
		goto error_exit;
	}

	/* --------------------------------------------- */

#ifdef DISPLAY_TIMESTAMPS
	prev_ts = linux_read_counter();
#endif

	for(i = 0; i < *iterations; i += (grid_dim * num_threads * 2)) {
		dim3 block_dimension(num_threads);
		dim3 grid_dimension(grid_dim);
		u32 process_amount = *iterations - i;
		u32 j;
		u32 match_count = 0;

		/* Determine the amount of keys that we */
		/* need to process on this pass through */
		/* the for() loop.                      */
		if(process_amount > (grid_dim * num_threads * 2)) {
			process_amount = (grid_dim * num_threads * 2);
		}

		/* Clear the match_found variable */
		if( hipMemset(cuda_match_found, 0, sizeof(u8)) != (hipError_t) hipSuccess ) {
			retval = -1;
			fprintf(stderr, "RC5 cuda: ERROR: hipMemset: cuda_match_found\r\n");
			goto error_exit;
		}

		/* Clear the results array */
		if( hipMemset(cuda_results, 0, grid_dim * num_threads * 2 * sizeof(u8)) != (hipError_t) hipSuccess ) {
			retval = -1;
			fprintf(stderr, "RC5 cuda: ERROR: hipMemset: cuda_results\r\n");
			goto error_exit;
		}

#ifdef DISPLAY_TIMESTAMPS
		current_ts = linux_read_counter();
		fprintf(stderr, "RC5 cuda: elapsed_time_1=%lli\r\n", current_ts - prev_ts);
		prev_ts = current_ts;
#endif

		/* Execute the CUDA core */
                
		cuda_2pipe<<<grid_dimension, block_dimension>>>(rc5_72unitwork->plain.hi, rc5_72unitwork->plain.lo,
		                                            rc5_72unitwork->cypher.hi, rc5_72unitwork->cypher.lo,
							    rc5_72unitwork->L0.hi, rc5_72unitwork->L0.mid, rc5_72unitwork->L0.lo,
		                                            (process_amount+1)/2, cuda_results, cuda_match_found);
		/* (process_amount+1)/2 just to handle the (impossible) case that process_amount is odd */
		{
			hipError_t last_error = hipGetLastError();
			if(last_error != (hipError_t) hipSuccess) {
				retval = -1;
				fprintf(stderr, "RC5 cuda: CUDA CORE ERROR: %s\r\n", hipGetErrorString(last_error));
				goto error_exit;
			}
		}

#ifdef DISPLAY_TIMESTAMPS
		current_ts = linux_read_counter();
		fprintf(stderr, "RC5 cuda: elapsed_time_2=%lli\r\n", current_ts - prev_ts);
		prev_ts = current_ts;
#endif

		/* Copy the match_found variable to the host */
		if( hipMemcpy((void *)&match_found, (void *)cuda_match_found, sizeof(u8), hipMemcpyDeviceToHost) != (hipError_t) hipSuccess ) {
			retval = -1;
			fprintf(stderr, "RC5 cuda: ERROR: hipMemcpy: cuda_match_found\r\n");
			goto error_exit;
		}

#ifdef DISPLAY_TIMESTAMPS
		current_ts = linux_read_counter();
		fprintf(stderr, "RC5 cuda: elapsed_time_3=%lli\r\n", current_ts - prev_ts);
		prev_ts = current_ts;
#endif

		/* Optimization: Only scan the results[] if  */
		/* the match_found flag (indicating an exact */
		/* or partial match) is set.                 */
		if(match_found) {
			
			/* Copy the results[] array to the host */
			if( hipMemcpy((void *)results, (void *)cuda_results, process_amount * sizeof(u8), hipMemcpyDeviceToHost) != (hipError_t) hipSuccess ) {
				retval = -1;
				fprintf(stderr, "RC5 cuda: ERROR: hipMemcpy: cuda_results\r\n");
				goto error_exit;
			}

			/* Check the results array for any matches. */
			for(j = 0; j < process_amount; j++) {

				/* Check if we have found a partial match */
				if(results[j] > 0) {
					rc5_72unitwork->check.count++;
					match_count++;

					/* Copy over the current key */
					rc5_72unitwork->check.hi = rc5_72unitwork->L0.hi;
					rc5_72unitwork->check.mid = rc5_72unitwork->L0.mid;
					rc5_72unitwork->check.lo = rc5_72unitwork->L0.lo;

					/* Offset the key index to match out current position */
					increment_L0(&rc5_72unitwork->check.hi, &rc5_72unitwork->check.mid, &rc5_72unitwork->check.lo, j);

					/* Check if we have found an exact match */
					if(results[j] > 1) {
						/* Correct the L0 offste value */
						increment_L0(&rc5_72unitwork->L0.hi, &rc5_72unitwork->L0.mid, &rc5_72unitwork->L0.lo, j);

						/* Pass back the iterations count to the callee */
						*iterations = i + j;

						/* Update the return value and jump to the exit point */
						retval = RESULT_FOUND;
						goto sucess_exit;
					}
				}

			} /* for(... j < process_amount ...) */

		} /* if(match_found) */

		/* Advance L0 by the amount that we processed */
		/* this pass through the for() loop.          */
		increment_L0(&rc5_72unitwork->L0.hi, &rc5_72unitwork->L0.mid, &rc5_72unitwork->L0.lo, process_amount);

#ifdef DISPLAY_TIMESTAMPS
		current_ts = linux_read_counter();
		fprintf(stderr, "RC5 cuda: elapsed_time_4=%lli\r\n", current_ts - prev_ts);
		prev_ts = current_ts;
#endif
	}

sucess_exit:
error_exit:
#ifdef DISPLAY_TIMESTAMPS
		current_ts = linux_read_counter();
		fprintf(stderr, "RC5 cuda: elapsed_time_5=%lli\r\n", current_ts - prev_ts);
		prev_ts = current_ts;
#endif

	if(cuda_match_found) {
		hipFree(cuda_match_found);
	}

	if(cuda_results) {
		hipFree(cuda_results);
	}

	if(results) {
		free(results);
	}

	if(retval == -1) {
		hipError_t last_error = hipGetLastError();
		fprintf(stderr, "RC5 cuda: error_exit\r\n");
		fprintf(stderr, "RC5 cuda: ERROR: %s\r\n", hipGetErrorString(last_error));
		fflush(stderr);
	}

#ifdef DISPLAY_TIMESTAMPS
		current_ts = linux_read_counter();
		fprintf(stderr, "RC5 cuda: elapsed_time_6=%lli\r\n", current_ts - prev_ts);
		prev_ts = current_ts;
#endif

	return retval;
}



/* Linux Only: Return the current uSec count */
#ifdef DISPLAY_TIMESTAMPS
static __inline int64_t linux_read_counter(void)
{
        struct timeval tv;
        int64_t retval = 0;

        gettimeofday(&tv, NULL);

        retval = (((int64_t)tv.tv_sec) * 1000000) + tv.tv_usec;

        return retval;
}
#endif /* ifdef DISPLAY_TIMESTAMPS */


/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------                  GPU Core                  --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

__global__ void cuda_2pipe(const u32 plain_hi, const u32 plain_lo,
                          const u32 cypher_hi, const u32 cypher_lo,
		          const u32 L0_hi, const u32 L0_mid, const u32 L0_lo,
                          const u32 process_amount, u8 * results, u8 * match_found)
{
	/* Grid of blocks dimension */
//	int gd = gridDim.x;

	/* Block index */
	int bx = blockIdx.x;

	/* Block of threads dimension */
	int bd = blockDim.x;

	/* Thread index */
	int tx = threadIdx.x;

	/* RC5 local state variables */
	u32 A1, A2, B1, B2;
	u32 S1[26], S2[26];
	u32 L1[3], L2[3];

	/* Drop out early if we don't have any data to process */
	if( ((bx * bd) + tx) > process_amount) {
		/* This processes two extra keys at the end of a non-full block */
		/* But this takes no extra time so might as well */
		/* Warning... Make sure you DON'T use  */
		/* __syncthreads() anywhere after this */
		/* point in the core!!!                */
		return;
	}

	/* Initialize the S[] with constants */
#define KEY_INIT(i) S1[i] = S2[i] = P + i*Q;
	KEY_INIT(0);
	KEY_INIT(1);
	KEY_INIT(2);
	KEY_INIT(3);
	KEY_INIT(4);
	KEY_INIT(5);
	KEY_INIT(6);
	KEY_INIT(7);
	KEY_INIT(8);
	KEY_INIT(9);
	KEY_INIT(10);
	KEY_INIT(11);
	KEY_INIT(12);
	KEY_INIT(13);
	KEY_INIT(14);
	KEY_INIT(15);
	KEY_INIT(16);
	KEY_INIT(17);
	KEY_INIT(18);
	KEY_INIT(19);
	KEY_INIT(20);
	KEY_INIT(21);
	KEY_INIT(22);
	KEY_INIT(23);
	KEY_INIT(24);
	KEY_INIT(25);

	/* Initialize L0[] based on our block    */
	/* and thread index.                     */
	L1[2] = L2[2] = L0_hi;
	L1[1] = L2[1] = L0_mid;
	L1[0] = L2[0] = L0_lo;
	increment_L0(&L1[2], &L1[1], &L1[0], 2*((bx * bd) + tx));
	increment_L0(&L2[2], &L2[1], &L2[0], 2*((bx * bd) + tx) + 0x01);

	/* ------------------------------------- */
	/* ------------------------------------- */
	/* ------------------------------------- */

#define ROTL_BLOCK(i,j) ROTL_BLOCK_j##j (i)

#define ROTL_BLOCK_i0_j1 \
    S1[0] = ROTL3(S1[0]+(S1[25]+L1[0])); \
    S2[0] = ROTL3(S2[0]+(S2[25]+L2[0])); \
    L1[1] = ROTL(L1[1]+(S1[0]+L1[0]),(S1[0]+L1[0])); \
    L2[1] = ROTL(L2[1]+(S2[0]+L2[0]),(S2[0]+L2[0])); \

#define ROTL_BLOCK_i0_j2 \
    S1[0] = ROTL3(S1[0]+(S1[25]+L1[1])); \
    S2[0] = ROTL3(S2[0]+(S2[25]+L2[1])); \
    L1[2] = ROTL(L1[2]+(S1[0]+L1[1]),(S1[0]+L1[1])); \
    L2[2] = ROTL(L2[2]+(S2[0]+L2[1]),(S2[0]+L2[1])); \

#define ROTL_BLOCK_j0(i) \
    S1[i] = ROTL3(S1[i]+(S1[i-1]+L1[2])); \
    S2[i] = ROTL3(S2[i]+(S2[i-1]+L2[2])); \
    L1[0] = ROTL(L1[0]+(S1[i]+L1[2]),(S1[i]+L1[2])); \
    L2[0] = ROTL(L2[0]+(S2[i]+L2[2]),(S2[i]+L2[2])); \

#define ROTL_BLOCK_j1(i) \
    S1[i] = ROTL3(S1[i]+(S1[i-1]+L1[0])); \
    S2[i] = ROTL3(S2[i]+(S2[i-1]+L2[0])); \
    L1[1] = ROTL(L1[1]+(S1[i]+L1[0]),(S1[i]+L1[0])); \
    L2[1] = ROTL(L2[1]+(S2[i]+L2[0]),(S2[i]+L2[0])); \

#define ROTL_BLOCK_j2(i) \
    S1[i] = ROTL3(S1[i]+(S1[i-1]+L1[1])); \
    S2[i] = ROTL3(S2[i]+(S2[i-1]+L2[1])); \
    L1[2] = ROTL(L1[2]+(S1[i]+L1[1]),(S1[i]+L1[1])); \
    L2[2] = ROTL(L2[2]+(S2[i]+L2[1]),(S2[i]+L2[1])); \

	/* ---------- */

    S1[0] = ROTL3(S1[0]);
    S2[0] = ROTL3(S2[0]);
    L1[0] = ROTL(L1[0]+S1[0],S1[0]);
    L2[0] = ROTL(L2[0]+S2[0],S2[0]); 

	/* ---------- */

    ROTL_BLOCK(1,1);
    ROTL_BLOCK(2,2);
    ROTL_BLOCK(3,0);
    ROTL_BLOCK(4,1);
    ROTL_BLOCK(5,2);
    ROTL_BLOCK(6,0);
    ROTL_BLOCK(7,1);
    ROTL_BLOCK(8,2);
    ROTL_BLOCK(9,0);
    ROTL_BLOCK(10,1);
    ROTL_BLOCK(11,2);
    ROTL_BLOCK(12,0);
    ROTL_BLOCK(13,1);
    ROTL_BLOCK(14,2);
    ROTL_BLOCK(15,0);
    ROTL_BLOCK(16,1);
    ROTL_BLOCK(17,2);
    ROTL_BLOCK(18,0);
    ROTL_BLOCK(19,1);
    ROTL_BLOCK(20,2);
    ROTL_BLOCK(21,0);
    ROTL_BLOCK(22,1);
    ROTL_BLOCK(23,2);
    ROTL_BLOCK(24,0);
    ROTL_BLOCK(25,1);

	/* ---------- */

	ROTL_BLOCK_i0_j2;

	/* ---------- */

    ROTL_BLOCK(1,0);
    ROTL_BLOCK(2,1);
    ROTL_BLOCK(3,2);
    ROTL_BLOCK(4,0);
    ROTL_BLOCK(5,1);
    ROTL_BLOCK(6,2);
    ROTL_BLOCK(7,0);
    ROTL_BLOCK(8,1);
    ROTL_BLOCK(9,2);
    ROTL_BLOCK(10,0);
    ROTL_BLOCK(11,1);
    ROTL_BLOCK(12,2);
    ROTL_BLOCK(13,0);
    ROTL_BLOCK(14,1);
    ROTL_BLOCK(15,2);
    ROTL_BLOCK(16,0);
    ROTL_BLOCK(17,1);
    ROTL_BLOCK(18,2);
    ROTL_BLOCK(19,0);
    ROTL_BLOCK(20,1);
    ROTL_BLOCK(21,2);
    ROTL_BLOCK(22,0);
    ROTL_BLOCK(23,1);
    ROTL_BLOCK(24,2);
    ROTL_BLOCK(25,0);

	/* ---------- */

	ROTL_BLOCK_i0_j1;

	/* ---------- */

    ROTL_BLOCK(1,2);
    ROTL_BLOCK(2,0);
    ROTL_BLOCK(3,1);
    ROTL_BLOCK(4,2);
    ROTL_BLOCK(5,0);
    ROTL_BLOCK(6,1);
    ROTL_BLOCK(7,2);
    ROTL_BLOCK(8,0);
    ROTL_BLOCK(9,1);
    ROTL_BLOCK(10,2);
    ROTL_BLOCK(11,0);
    ROTL_BLOCK(12,1);
    ROTL_BLOCK(13,2);
    ROTL_BLOCK(14,0);
    ROTL_BLOCK(15,1);
    ROTL_BLOCK(16,2);
    ROTL_BLOCK(17,0);
    ROTL_BLOCK(18,1);
    ROTL_BLOCK(19,2);
    ROTL_BLOCK(20,0);
    ROTL_BLOCK(21,1);
    ROTL_BLOCK(22,2);
    ROTL_BLOCK(23,0);
    ROTL_BLOCK(24,1);
    ROTL_BLOCK(25,2);

	/* ---------- */

	A1 = plain_lo + S1[0];
	A2 = plain_lo + S2[0];
	B1 = plain_hi + S1[1];
	B2 = plain_hi + S2[1];

	/* ---------- */

#define FINAL_BLOCK(i) \
    A1 = ROTL(A1^B1,B1)+S1[2*i]; \
    A2 = ROTL(A2^B2,B2)+S2[2*i]; \
    B1 = ROTL(B1^A1,A1)+S1[2*i+1]; \
    B2 = ROTL(B2^A2,A2)+S2[2*i+1];

    FINAL_BLOCK(1);
    FINAL_BLOCK(2);
    FINAL_BLOCK(3);
    FINAL_BLOCK(4);
    FINAL_BLOCK(5);
    FINAL_BLOCK(6);
    FINAL_BLOCK(7);
    FINAL_BLOCK(8);
    FINAL_BLOCK(9);
    FINAL_BLOCK(10);
    FINAL_BLOCK(11);
    FINAL_BLOCK(12);

	/* ------------------------------------- */
	/* ------------------------------------- */
	/* ------------------------------------- */

	/* Check the results for a match.        */
	if (A1 == cypher_lo) {

		/* Set the match_found flag */
		*match_found = 1;

		/* Record the "check_*" match   */
		/* in the results array.        */
		results[2*((bx * bd) + tx)] = 1;

		if (B1 == cypher_hi) {
			/* Record the RESULT_FOUND match  */
			/* in the results array.          */
			results[2*((bx * bd) + tx)] = 2;
		}
	}

        if (A2 == cypher_lo) {

		/* Set the match_found flag */
		*match_found = 1;

		/* Record the "check_*" match   */
		/* in the results array.        */
		results[2*((bx * bd) + tx)+1] = 1;

		if (B2 == cypher_hi) {
			/* Record the RESULT_FOUND match  */
			/* in the results array.          */
			results[2*((bx * bd) + tx)+1] = 2;
		}
	}
}

// vim: syntax=cpp
