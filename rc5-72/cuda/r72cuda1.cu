#include "hip/hip_runtime.h"
/*
 * Copyright Paul Kurucz 2007 - All Rights Reserved
 * For use in distributed.net projects only.
 * Any other distribution or use of this source violates copyright.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "ccoreio.h"

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

/* Uncomment the define below to display the    */
/* processing timestamps.  (Linux Only)         */
//#define DISPLAY_TIMESTAMPS

#ifdef DISPLAY_TIMESTAMPS
#include <sys/time.h>
#endif

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

const char *r72cuda1_cu(void) {
        return "@(#)$Id: r72cuda1.cu,v 1.1 2008/11/22 07:58:51 jlawson Exp $";
}

#define P 0xB7E15163
#define Q 0x9E3779B9

#ifdef __cplusplus
extern "C" s32 CDECL rc5_72_unit_func_cuda_1( RC5_72UnitWork *, u32 *, void * );
#endif

/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------              Local Variables               --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

#if 0
typedef struct
{
        struct {u32 hi,lo;} plain;  /* plaintext (already mixed with iv!) */
        struct {u32 hi,lo;} cypher; /* cyphertext */
        struct {u32 hi,mid,lo;} L0; /* key, changes with every unit *
PIPELINE_COUNT. */
        struct {u32 count; u32 hi,mid,lo;} check; /* counter-measure check */
} DNETC_PACKED RC5_72UnitWork;
#endif

/* Type decaration for the L0 field of the      */
/* RC5_72UnitWork structure.                    */
typedef struct {
        u32 hi;
        u32 mid;
        u32 lo;
} L0_t;

/* The number of GPU threads per thread block   */
/* to execute.  The default value of 64 makes   */
/* optimum usage of __shared__ multiprocessor   */
/* memory.  The maximum value is 512.           */
const u32 num_threads = 96;


/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------     Local Helper Function Prototypes       --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

static __host__ __device__ u32 swap_u32(u32 num);

static __host__ __device__ u8 add_u32(u32 num1, u32 num2, u32 * result);

static __host__ __device__ void increment_L0(u32 * hi, u32 * mid, u32* lo, u32 amount);

static __global__ void cuda_core(const u32 plain_hi, const u32 plain_lo,
                                 const u32 cypher_hi, const u32 cypher_lo,
                                 const u32 L0_hi, const u32 L0_mid, const u32 L0_lo,
                                 const u32 process_amount, u8 *results, u8 * match_found);

#ifdef DISPLAY_TIMESTAMPS
static __inline int64_t linux_read_counter(void);
#endif

#define SHL(x, s) ((u32) ((x) << ((s) & 31)))
#define SHR(x, s) ((u32) ((x) >> (32 - ((s) & 31))))
#define ROTL(x, s) ((u32) (SHL((x), (s)) | SHR((x), (s))))
#define ROTL3(x) ROTL(x, 3)


/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------              Core Entry Point              --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

s32 CDECL rc5_72_unit_func_cuda_1(RC5_72UnitWork *rc5_72unitwork, u32
*iterations, void * /*memblk*/)
{
        u32 i;
        u32 grid_dim;
        s32 retval = RESULT_NOTHING;

        /* Local and GPU variable pairs */
        u8 match_found;
        u8 * cuda_match_found = NULL;
        u8 * results = NULL;
        u8 * cuda_results = NULL;

#ifdef DISPLAY_TIMESTAMPS
        int64_t current_ts;
        int64_t prev_ts;
#endif

//      fprintf(stderr, "\r\nRC5 cuda: thread=%i, iterations=%i\r\n",rc5_72unitwork->threadnum, *iterations);

        hipSetDevice(rc5_72unitwork->threadnum);

        /* Determine the grid dimensionality based on the */
        /* number of iterations.                          */
        grid_dim = (*iterations / num_threads) + 1;
        if(grid_dim > 65535) {
                grid_dim = 65535;
        }

        /* --------------------------------------------- */

        /* Allocate the cuda_match_found variable */
        if( hipMalloc((void **)&cuda_match_found, sizeof(u8)) != hipSuccess ) {
                retval = -1;
                fprintf(stderr, "RC5 cuda: ERROR: hipMalloc: cuda_match_found\r\n");
                goto error_exit;
        }

        /* Allocate the results arrays */
        results = (u8 *)malloc(grid_dim * num_threads);
        if( results == NULL ) {
                retval = -1;
                fprintf(stderr, "RC5 cuda: ERROR: malloc\r\n");
                goto error_exit;
        }

        if( hipMalloc((void **)&cuda_results, grid_dim * num_threads) != hipSuccess ) {
                retval = -1;
                fprintf(stderr, "RC5 cuda: ERROR: hipMalloc: cuda_results\r\n");
                goto error_exit;
        }

        /* --------------------------------------------- */

#ifdef DISPLAY_TIMESTAMPS
        prev_ts = linux_read_counter();
#endif

        for(i = 0; i < *iterations; i += (grid_dim * num_threads)) {
                dim3 block_dimension(num_threads);
                dim3 grid_dimension(grid_dim);
                u32 process_amount = *iterations - i;
                u32 j;
                u32 match_count = 0;

                /* Determine the amount of keys that we */
                /* need to process on this pass through */
                /* the for() loop.                      */
                if(process_amount > (grid_dim * num_threads)) {
                        process_amount = (grid_dim * num_threads);
                }

                /* Clear the match_found variable */
                if( hipMemset(cuda_match_found, 0, sizeof(u8)) != hipSuccess ) {
                        retval = -1;
                        fprintf(stderr, "RC5 cuda: ERROR: hipMemset: cuda_match_found\r\n");
                        goto error_exit;
                }

                /* Clear the results array */
                if( hipMemset(cuda_results, 0, grid_dim * num_threads) != hipSuccess ) {
                        retval = -1;
                        fprintf(stderr, "RC5 cuda: ERROR: hipMemset: cuda_results\r\n");
                        goto error_exit;
                }

#ifdef DISPLAY_TIMESTAMPS
                current_ts = linux_read_counter();
                fprintf(stderr, "RC5 cuda: elapsed_time_1=%lli\r\n", current_ts - prev_ts);
                prev_ts = current_ts;
#endif

                /* Execute the CUDA core */
                cuda_core<<<grid_dimension, block_dimension>>>(
			rc5_72unitwork->plain.hi, rc5_72unitwork->plain.lo,
			rc5_72unitwork->cypher.hi, rc5_72unitwork->cypher.lo,
			rc5_72unitwork->L0.hi, rc5_72unitwork->L0.mid, rc5_72unitwork->L0.lo,
			process_amount, cuda_results, cuda_match_found);
                {
                        hipError_t last_error = hipGetLastError();
                        if(last_error != hipSuccess) {
                                retval = -1;
                                fprintf(stderr, "RC5 cuda: CUDA CORE ERROR: %s\r\n", hipGetErrorString(last_error));
                                goto error_exit;
                        }
                }

#ifdef DISPLAY_TIMESTAMPS
                current_ts = linux_read_counter();
                fprintf(stderr, "RC5 cuda: elapsed_time_2=%lli\r\n", current_ts - prev_ts);
                prev_ts = current_ts;
#endif

                /* Copy the match_found variable to the host */
                if( hipMemcpy((void *)&match_found, (void*)cuda_match_found, sizeof(u8), hipMemcpyDeviceToHost) != hipSuccess ) {
                        retval = -1;
                        fprintf(stderr, "RC5 cuda: ERROR: hipMemcpy: cuda_match_found\r\n");
                        goto error_exit;
                }

#ifdef DISPLAY_TIMESTAMPS
                current_ts = linux_read_counter();
                fprintf(stderr, "RC5 cuda: elapsed_time_3=%lli\r\n", current_ts - prev_ts);
                prev_ts = current_ts;
#endif

                /* Optimization: Only scan the results[] if  */
                /* the match_found flag (indicating an exact */
                /* or partial match) is set.                 */
                if(match_found) {

                        /* Copy the results[] array to the host */
                        if( hipMemcpy((void *)results, (void*)cuda_results, process_amount, hipMemcpyDeviceToHost) != hipSuccess ) {
                                retval = -1;
                                fprintf(stderr, "RC5 cuda: ERROR: hipMemcpy: cuda_results\r\n");
                                goto error_exit;
                        }

                        /* Check the results array for any matches. */
                        for(j = 0; j < process_amount; j++) {

                                /* Check if we have found a partial match */
                                if(results[j] > 0) {
                                        rc5_72unitwork->check.count++;
                                        match_count++;

                                        /* Copy over the current key */
                                        rc5_72unitwork->check.hi = rc5_72unitwork->L0.hi;
                                        rc5_72unitwork->check.mid = rc5_72unitwork->L0.mid;
                                        rc5_72unitwork->check.lo = rc5_72unitwork->L0.lo;

                                        /* Offset the key index to match out current position */

					increment_L0(&rc5_72unitwork->check.hi, &rc5_72unitwork->check.mid, &rc5_72unitwork->check.lo, j);

                                        /* Check if we have found an exact match */
                                        if(results[j] > 1) {
                                                /* Correct the L0 offste value */

						increment_L0(&rc5_72unitwork->L0.hi, &rc5_72unitwork->L0.mid, &rc5_72unitwork->L0.lo, j);

                                                /* Pass back the iterations count to the callee */
                                                *iterations = i + j;

                                                /* Update the return value and jump to the exit point */
                                                retval = RESULT_FOUND;
                                                goto sucess_exit;
                                        }
                                }

                        } /* for(... j < process_amount ...) */

                } /* if(match_found) */

                /* Advance L0 by the amount that we processed */
                /* this pass through the for() loop.          */
                increment_L0(&rc5_72unitwork->L0.hi, &rc5_72unitwork->L0.mid, &rc5_72unitwork->L0.lo, process_amount);

#ifdef DISPLAY_TIMESTAMPS
                current_ts = linux_read_counter();
                fprintf(stderr, "RC5 cuda: elapsed_time_4=%lli\r\n", current_ts - prev_ts);
                prev_ts = current_ts;
#endif
        }

sucess_exit:
error_exit:
#ifdef DISPLAY_TIMESTAMPS
                current_ts = linux_read_counter();
                fprintf(stderr, "RC5 cuda: elapsed_time_5=%lli\r\n", current_ts - prev_ts);
                prev_ts = current_ts;
#endif

        if(cuda_match_found) {
                hipFree(cuda_match_found);
        }

        if(cuda_results) {
                hipFree(cuda_results);
        }

        if(results) {
                free(results);
        }

        if(retval == -1) {
                hipError_t last_error = hipGetLastError();
                fprintf(stderr, "RC5 cuda: error_exit\r\n");
                fprintf(stderr, "RC5 cuda: ERROR: %s\r\n", hipGetErrorString(last_error));
                fflush(stderr);
        }

#ifdef DISPLAY_TIMESTAMPS
                current_ts = linux_read_counter();
                fprintf(stderr, "RC5 cuda: elapsed_time_6=%lli\r\n", current_ts - prev_ts);
                prev_ts = current_ts;
#endif

//      fprintf(stderr, "RC5 cuda: thread=%i EXIT\r\n",rc5_72unitwork->threadnum);

        return retval;
}


/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------           Local Helper Functions           --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

/* u32 byte swap */
static __host__ __device__ u32 swap_u32(u32 num)
{
        u32 retval = (num & 0xFF000000) >> 24;
        retval |= (num & 0x00FF0000) >> 8;
        retval |= (num & 0x0000FF00) << 8;
        retval |= (num & 0x000000FF) << 24;

        return retval;
}

/* Adds two u32s, returning the carry out bit.  */
static __host__ __device__ u8 add_u32(u32 num1, u32 num2, u32 * result)
{
        u8 carry = 0;
        u32 temp = num1;

        temp += num2;

        /* Check for an overflow */
        if(temp < num1) {
                carry = 1;
        }

        /* Pass back the result */
        *result = temp;

        return carry;
}

/* Increments the hi, mid and lo parts of the   */
/* L0 by the specified amount.                  */
static __host__ __device__ void increment_L0(u32 * hi, u32 * mid, u32
* lo, u32 amount)
{
        u32 temp;
        u32 result;
        u8 carry;

        /* Low uint32 */
        temp = *hi & 0xFF;
        temp |= swap_u32(*mid) << 8;
        carry = add_u32(temp, amount, &result);
        *hi = result & 0xFF;
        *mid &= 0x000000FF;
        *mid |= swap_u32(result >> 8);

        /* Mid uint32 */
        if(carry) {
                temp = *mid & 0xFF;
                temp |= swap_u32(*lo) << 8;
                carry = add_u32(temp, 1, &result);
                *mid &= 0xFFFFFF00;
                *mid |= result & 0xFF;
                *lo &= 0x000000FF;
                *lo |= swap_u32(result >> 8);
        }

        if(carry) {
                temp = *lo & 0xFF;
                carry = add_u32(temp, 1, &result);
                *lo &= 0xFFFFFF00;
                *lo |= result & 0xFF;
        }
}

/* Linux Only: Return the current uSec count */
#ifdef DISPLAY_TIMESTAMPS
static __inline int64_t linux_read_counter(void)
{
        struct timeval tv;
        int64_t retval = 0;

        gettimeofday(&tv, NULL);

        retval = (((int64_t)tv.tv_sec) * 1000000) + tv.tv_usec;

        return retval;
}
#endif /* ifdef DISPLAY_TIMESTAMPS */


/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */
/* ---------------                  GPU Core                  --------------- */
/* -------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------- */

__global__ void cuda_core(const u32 plain_hi, const u32 plain_lo,
                          const u32 cypher_hi, const u32 cypher_lo,
                          const u32 L0_hi, const u32 L0_mid, const u32 L0_lo,
                          const u32 process_amount, u8 * results, u8 *match_found)
{
        /* Grid of blocks dimension */
//      int gd = gridDim.x;

        /* Block index */
        int bx = blockIdx.x;

        /* Block of threads dimension */
        int bd = blockDim.x;

        /* Thread index */
        int tx = threadIdx.x;

        /* RC5 local state variables */
        u32 A, B;
        u32 S0, S1, S2, S3, S4, S5, S6, S7, S8, S9, S10, S11, S12,
          S13, S14, S15, S16, S17, S18, S19, S20, S21, S22, S23, S24, S25;
        u32 L[3];

        /* Drop out early if we don't have any data to process */
        if( ((bx * bd) + tx) > process_amount) {
                /* Warning... Make sure you DON'T use  */
                /* __syncthreads() anywhere after this */
                /* point in the core!!!                */
                return;
        }

        /* Initialize the S[] with constants */
#define KEY_INIT(_i) S##_i = (P + _i*Q)
        KEY_INIT(0);
        KEY_INIT(1);
        KEY_INIT(2);
        KEY_INIT(3);
        KEY_INIT(4);
        KEY_INIT(5);
        KEY_INIT(6);
        KEY_INIT(7);
        KEY_INIT(8);
        KEY_INIT(9);
        KEY_INIT(10);
        KEY_INIT(11);
        KEY_INIT(12);
        KEY_INIT(13);
        KEY_INIT(14);
        KEY_INIT(15);
        KEY_INIT(16);
        KEY_INIT(17);
        KEY_INIT(18);
        KEY_INIT(19);
        KEY_INIT(20);
        KEY_INIT(21);
        KEY_INIT(22);
        KEY_INIT(23);
        KEY_INIT(24);
        KEY_INIT(25);

        /* Initialize L0[] based on our block    */
        /* and thread index.                     */
        L[2] = L0_hi;
        L[1] = L0_mid;
        L[0] = L0_lo;
        increment_L0(&L[2], &L[1], &L[0], (bx * bd) + tx);

        /* ------------------------------------- */
        /* ------------------------------------- */
        /* ------------------------------------- */

#define ROTL_BLOCK_k(i, j, k) ROTL_BLOCK_k##k (i, j)

#define ROTL_BLOCK_k0(i, j) \
        S##i = ROTL3(S##i+(S##j+L[2])); \
        L[0] = ROTL(L[0]+(S##i+L[2]),(S##i+L[2])); \

#define ROTL_BLOCK_k1(i, j) \
        S##i = ROTL3(S##i+(S##j+L[0])); \
        L[1] = ROTL(L[1]+(S##i+L[0]),(S##i+L[0])); \

#define ROTL_BLOCK_k2(i, j) \
        S##i = ROTL3(S##i+(S##j+L[1])); \
        L[2] = ROTL(L[2]+(S##i+L[1]),(S##i+L[1])); \

#define ROTL_BLOCK_i0_j1 \
        S0 = ROTL3(S0+(S25+L[0])); \
        L[1] = ROTL(L[1]+(S0+L[0]),(S0+L[0])); \

#define ROTL_BLOCK_i0_j2 \
        S0 = ROTL3(S0+(S25+L[1])); \
        L[2] = ROTL(L[2]+(S0+L[1]),(S0+L[1])); \

        /* ---------- */

        S0 = ROTL3(S0);
        L[0] = ROTL(L[0]+S0,S0);

        /* ---------- */

        ROTL_BLOCK_k(1, 0, 1);
        ROTL_BLOCK_k(2, 1, 2);
        ROTL_BLOCK_k(3, 2, 0);
        ROTL_BLOCK_k(4, 3, 1);
        ROTL_BLOCK_k(5, 4, 2);
        ROTL_BLOCK_k(6, 5, 0);
        ROTL_BLOCK_k(7, 6, 1);
        ROTL_BLOCK_k(8, 7, 2);
        ROTL_BLOCK_k(9, 8, 0);
        ROTL_BLOCK_k(10, 9, 1);
        ROTL_BLOCK_k(11, 10, 2);
        ROTL_BLOCK_k(12, 11, 0);
        ROTL_BLOCK_k(13, 12, 1);
        ROTL_BLOCK_k(14, 13, 2);
        ROTL_BLOCK_k(15, 14, 0);
        ROTL_BLOCK_k(16, 15, 1);
        ROTL_BLOCK_k(17, 16, 2);
        ROTL_BLOCK_k(18, 17, 0);
        ROTL_BLOCK_k(19, 18, 1);
        ROTL_BLOCK_k(20, 19, 2);
        ROTL_BLOCK_k(21, 20, 0);
        ROTL_BLOCK_k(22, 21, 1);
        ROTL_BLOCK_k(23, 22, 2);
        ROTL_BLOCK_k(24, 23, 0);
        ROTL_BLOCK_k(25, 24, 1);

        /* ---------- */

        ROTL_BLOCK_i0_j2;

        /* ---------- */

        ROTL_BLOCK_k(1, 0, 0);
        ROTL_BLOCK_k(2, 1, 1);
        ROTL_BLOCK_k(3, 2, 2);
        ROTL_BLOCK_k(4, 3, 0);
        ROTL_BLOCK_k(5, 4, 1);
        ROTL_BLOCK_k(6, 5, 2);
        ROTL_BLOCK_k(7, 6, 0);
        ROTL_BLOCK_k(8, 7, 1);
        ROTL_BLOCK_k(9, 8, 2);
        ROTL_BLOCK_k(10, 9, 0);
        ROTL_BLOCK_k(11, 10, 1);
        ROTL_BLOCK_k(12, 11, 2);
        ROTL_BLOCK_k(13, 12, 0);
        ROTL_BLOCK_k(14, 13, 1);
        ROTL_BLOCK_k(15, 14, 2);
        ROTL_BLOCK_k(16, 15, 0);
        ROTL_BLOCK_k(17, 16, 1);
        ROTL_BLOCK_k(18, 17, 2);
        ROTL_BLOCK_k(19, 18, 0);
        ROTL_BLOCK_k(20, 19, 1);
        ROTL_BLOCK_k(21, 20, 2);
        ROTL_BLOCK_k(22, 21, 0);
        ROTL_BLOCK_k(23, 22, 1);
        ROTL_BLOCK_k(24, 23, 2);
        ROTL_BLOCK_k(25, 24, 0);

        /* ---------- */

        ROTL_BLOCK_i0_j1;

        /* ---------- */

        ROTL_BLOCK_k(1, 0, 2);
        ROTL_BLOCK_k(2, 1, 0);
        ROTL_BLOCK_k(3, 2, 1);
        ROTL_BLOCK_k(4, 3, 2);
        ROTL_BLOCK_k(5, 4, 0);
        ROTL_BLOCK_k(6, 5, 1);
        ROTL_BLOCK_k(7, 6, 2);
        ROTL_BLOCK_k(8, 7, 0);
        ROTL_BLOCK_k(9, 8, 1);
        ROTL_BLOCK_k(10, 9, 2);
        ROTL_BLOCK_k(11, 10, 0);
        ROTL_BLOCK_k(12, 11, 1);
        ROTL_BLOCK_k(13, 12, 2);
        ROTL_BLOCK_k(14, 13, 0);
        ROTL_BLOCK_k(15, 14, 1);
        ROTL_BLOCK_k(16, 15, 2);
        ROTL_BLOCK_k(17, 16, 0);
        ROTL_BLOCK_k(18, 17, 1);
        ROTL_BLOCK_k(19, 18, 2);
        ROTL_BLOCK_k(20, 19, 0);
        ROTL_BLOCK_k(21, 20, 1);
        ROTL_BLOCK_k(22, 21, 2);
        ROTL_BLOCK_k(23, 22, 0);
        ROTL_BLOCK_k(24, 23, 1);
        ROTL_BLOCK_k(25, 24, 2);

        /* ---------- */

        A = plain_lo + S0;
        B = plain_hi + S1;

        /* ---------- */

#define FINAL_BLOCK_k(i, j) \
        A = ROTL(A^B,B)+S##i; \
        B = ROTL(B^A,A)+S##j;

        FINAL_BLOCK_k(2, 3);
        FINAL_BLOCK_k(4, 5);
        FINAL_BLOCK_k(6, 7);
        FINAL_BLOCK_k(8, 9);
        FINAL_BLOCK_k(10, 11);
        FINAL_BLOCK_k(12, 13);
        FINAL_BLOCK_k(14, 15);
        FINAL_BLOCK_k(16, 17);
        FINAL_BLOCK_k(18, 19);
        FINAL_BLOCK_k(20, 21);
        FINAL_BLOCK_k(22, 23);
        FINAL_BLOCK_k(24, 25);

        /* ------------------------------------- */
        /* ------------------------------------- */
        /* ------------------------------------- */

        /* Check the results for a match.        */
        if (A == cypher_lo) {

                /* Set the match_found flag */
                *match_found = 1;

                /* Record the "check_*" match   */
                /* in the results array.        */
                results[(bx * bd) + tx] = 1;

                if (B == cypher_hi) {
                        /* Record the RESULT_FOUND match  */
                        /* in the results array.          */
                        results[(bx * bd) + tx] = 2;
                }
        }
}
